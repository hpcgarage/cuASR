/***************************************************************************************************
* Copyright (c) 2022, Vijay Thakkar (thakkarv@gatech.edu).
**************************************************************************************************/
//////////////////////////////////////////////////////////////////////
//  THIS BENCHMARK FILE IS GENERATED AUTOMATICALLY : DO NOT MODIFY  //
//////////////////////////////////////////////////////////////////////

#include "benchmark/benchmark.h"

#include "cuasr/gemm/device/default_srgemm_configuration.h"
#include "cuasr/gemm/device/srgemm.h"
#include "cuasr/functional.h"

#include "harness.h"

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_plus_mult_f64_srgemm_tt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::plus_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_plus_mult_f64_srgemm_tt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_plus_mult_f64_srgemm_tt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::plus_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_plus_mult_f64_srgemm_tt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_plus_mult_f64_srgemm_tn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::plus_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_plus_mult_f64_srgemm_tn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_plus_mult_f64_srgemm_tn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::plus_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_plus_mult_f64_srgemm_tn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_plus_mult_f64_srgemm_nt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::plus_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_plus_mult_f64_srgemm_nt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_plus_mult_f64_srgemm_nt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::plus_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_plus_mult_f64_srgemm_nt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_plus_mult_f64_srgemm_nn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::plus_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_plus_mult_f64_srgemm_nn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_plus_mult_f64_srgemm_nn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::plus_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_plus_mult_f64_srgemm_nn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_plus_mult_f32_srgemm_tt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::plus_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_plus_mult_f32_srgemm_tt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_plus_mult_f32_srgemm_tt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::plus_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_plus_mult_f32_srgemm_tt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_plus_mult_f32_srgemm_tn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::plus_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_plus_mult_f32_srgemm_tn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_plus_mult_f32_srgemm_tn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::plus_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_plus_mult_f32_srgemm_tn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_plus_mult_f32_srgemm_nt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::plus_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_plus_mult_f32_srgemm_nt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_plus_mult_f32_srgemm_nt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::plus_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_plus_mult_f32_srgemm_nt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_plus_mult_f32_srgemm_nn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::plus_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_plus_mult_f32_srgemm_nn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_plus_mult_f32_srgemm_nn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::plus_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_plus_mult_f32_srgemm_nn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_plus_f64_srgemm_tt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_plus_f64_srgemm_tt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_plus_f64_srgemm_tt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_plus_f64_srgemm_tt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_plus_f64_srgemm_tn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_plus_f64_srgemm_tn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_plus_f64_srgemm_tn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_plus_f64_srgemm_tn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_plus_f64_srgemm_nt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_plus_f64_srgemm_nt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_plus_f64_srgemm_nt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_plus_f64_srgemm_nt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_plus_f64_srgemm_nn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_plus_f64_srgemm_nn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_plus_f64_srgemm_nn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_plus_f64_srgemm_nn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_plus_f32_srgemm_tt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_plus_f32_srgemm_tt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_plus_f32_srgemm_tt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_plus_f32_srgemm_tt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_plus_f32_srgemm_tn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_plus_f32_srgemm_tn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_plus_f32_srgemm_tn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_plus_f32_srgemm_tn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_plus_f32_srgemm_nt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_plus_f32_srgemm_nt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_plus_f32_srgemm_nt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_plus_f32_srgemm_nt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_plus_f32_srgemm_nn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_plus_f32_srgemm_nn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_plus_f32_srgemm_nn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_plus_f32_srgemm_nn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_plus_f64_srgemm_tt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_plus_f64_srgemm_tt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_plus_f64_srgemm_tt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_plus_f64_srgemm_tt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_plus_f64_srgemm_tn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_plus_f64_srgemm_tn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_plus_f64_srgemm_tn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_plus_f64_srgemm_tn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_plus_f64_srgemm_nt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_plus_f64_srgemm_nt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_plus_f64_srgemm_nt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_plus_f64_srgemm_nt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_plus_f64_srgemm_nn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_plus_f64_srgemm_nn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_plus_f64_srgemm_nn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_plus_f64_srgemm_nn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_plus_f32_srgemm_tt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_plus_f32_srgemm_tt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_plus_f32_srgemm_tt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_plus_f32_srgemm_tt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_plus_f32_srgemm_tn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_plus_f32_srgemm_tn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_plus_f32_srgemm_tn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_plus_f32_srgemm_tn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_plus_f32_srgemm_nt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_plus_f32_srgemm_nt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_plus_f32_srgemm_nt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_plus_f32_srgemm_nt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_plus_f32_srgemm_nn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_plus_f32_srgemm_nn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_plus_f32_srgemm_nn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_plus<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_plus_f32_srgemm_nn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_max_f64_srgemm_tt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_max<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_max_f64_srgemm_tt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_max_f64_srgemm_tt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_max<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_max_f64_srgemm_tt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_max_f64_srgemm_tn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_max<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_max_f64_srgemm_tn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_max_f64_srgemm_tn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_max<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_max_f64_srgemm_tn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_max_f64_srgemm_nt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_max<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_max_f64_srgemm_nt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_max_f64_srgemm_nt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_max<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_max_f64_srgemm_nt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_max_f64_srgemm_nn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_max<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_max_f64_srgemm_nn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_max_f64_srgemm_nn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_max<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_max_f64_srgemm_nn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_max_f32_srgemm_tt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_max<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_max_f32_srgemm_tt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_max_f32_srgemm_tt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_max<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_max_f32_srgemm_tt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_max_f32_srgemm_tn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_max<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_max_f32_srgemm_tn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_max_f32_srgemm_tn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_max<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_max_f32_srgemm_tn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_max_f32_srgemm_nt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_max<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_max_f32_srgemm_nt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_max_f32_srgemm_nt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_max<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_max_f32_srgemm_nt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_max_f32_srgemm_nn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_max<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_max_f32_srgemm_nn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_max_f32_srgemm_nn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_max<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_max_f32_srgemm_nn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_min_f64_srgemm_tt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_min<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_min_f64_srgemm_tt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_min_f64_srgemm_tt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_min<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_min_f64_srgemm_tt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_min_f64_srgemm_tn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_min<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_min_f64_srgemm_tn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_min_f64_srgemm_tn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_min<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_min_f64_srgemm_tn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_min_f64_srgemm_nt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_min<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_min_f64_srgemm_nt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_min_f64_srgemm_nt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_min<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_min_f64_srgemm_nt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_min_f64_srgemm_nn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_min<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_min_f64_srgemm_nn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_min_f64_srgemm_nn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_min<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_min_f64_srgemm_nn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_min_f32_srgemm_tt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_min<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_min_f32_srgemm_tt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_min_f32_srgemm_tt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_min<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_min_f32_srgemm_tt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_min_f32_srgemm_tn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_min<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_min_f32_srgemm_tn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_min_f32_srgemm_tn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_min<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_min_f32_srgemm_tn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_min_f32_srgemm_nt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_min<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_min_f32_srgemm_nt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_min_f32_srgemm_nt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_min<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_min_f32_srgemm_nt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_min_f32_srgemm_nn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_min<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_min_f32_srgemm_nn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_min_f32_srgemm_nn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_min<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_min_f32_srgemm_nn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_mult_f64_srgemm_tt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_mult_f64_srgemm_tt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_mult_f64_srgemm_tt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_mult_f64_srgemm_tt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_mult_f64_srgemm_tn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_mult_f64_srgemm_tn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_mult_f64_srgemm_tn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_mult_f64_srgemm_tn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_mult_f64_srgemm_nt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_mult_f64_srgemm_nt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_mult_f64_srgemm_nt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_mult_f64_srgemm_nt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_mult_f64_srgemm_nn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_mult_f64_srgemm_nn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_mult_f64_srgemm_nn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_mult_f64_srgemm_nn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_mult_f32_srgemm_tt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_mult_f32_srgemm_tt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_mult_f32_srgemm_tt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_mult_f32_srgemm_tt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_mult_f32_srgemm_tn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_mult_f32_srgemm_tn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_mult_f32_srgemm_tn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_mult_f32_srgemm_tn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_mult_f32_srgemm_nt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_mult_f32_srgemm_nt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_mult_f32_srgemm_nt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_mult_f32_srgemm_nt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_mult_f32_srgemm_nn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_mult_f32_srgemm_nn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_min_mult_f32_srgemm_nn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::min_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_min_mult_f32_srgemm_nn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_mult_f64_srgemm_tt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_mult_f64_srgemm_tt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_mult_f64_srgemm_tt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_mult_f64_srgemm_tt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_mult_f64_srgemm_tn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_mult_f64_srgemm_tn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_mult_f64_srgemm_tn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_mult_f64_srgemm_tn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_mult_f64_srgemm_nt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_mult_f64_srgemm_nt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_mult_f64_srgemm_nt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_mult_f64_srgemm_nt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_mult_f64_srgemm_nn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_mult_f64_srgemm_nn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_mult_f64_srgemm_nn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_mult_f64_srgemm_nn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_mult_f32_srgemm_tt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_mult_f32_srgemm_tt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_mult_f32_srgemm_tt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_mult_f32_srgemm_tt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_mult_f32_srgemm_tn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_mult_f32_srgemm_tn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_mult_f32_srgemm_tn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_mult_f32_srgemm_tn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_mult_f32_srgemm_nt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_mult_f32_srgemm_nt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_mult_f32_srgemm_nt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_mult_f32_srgemm_nt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_mult_f32_srgemm_nn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_mult_f32_srgemm_nn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_max_mult_f32_srgemm_nn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::max_mult<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_max_mult_f32_srgemm_nn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_or_and_f64_srgemm_tt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::or_and<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_or_and_f64_srgemm_tt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_or_and_f64_srgemm_tt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::or_and<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_or_and_f64_srgemm_tt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_or_and_f64_srgemm_tn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::or_and<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_or_and_f64_srgemm_tn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_or_and_f64_srgemm_tn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::or_and<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_or_and_f64_srgemm_tn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_or_and_f64_srgemm_nt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::or_and<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_or_and_f64_srgemm_nt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_or_and_f64_srgemm_nt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::or_and<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_or_and_f64_srgemm_nt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_or_and_f64_srgemm_nn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::or_and<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_or_and_f64_srgemm_nn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_or_and_f64_srgemm_nn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = double;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::or_and<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_or_and_f64_srgemm_nn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_or_and_f32_srgemm_tt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::or_and<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_or_and_f32_srgemm_tt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_or_and_f32_srgemm_tt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::or_and<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_or_and_f32_srgemm_tt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_or_and_f32_srgemm_tn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::or_and<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_or_and_f32_srgemm_tn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_or_and_f32_srgemm_tn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::or_and<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_or_and_f32_srgemm_tn_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_or_and_f32_srgemm_nt_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::or_and<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_or_and_f32_srgemm_nt_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_or_and_f32_srgemm_nt_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::or_and<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_or_and_f32_srgemm_nt_t)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_or_and_f32_srgemm_nn_n(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::or_and<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_or_and_f32_srgemm_nn_n)
    ->RangeMultiplier(2)->Range(256, 4096);

///////////////////////////////////////////////////////////////////////////////

static void BM_SM80_default_or_and_f32_srgemm_nn_t(benchmark::State &state) {
  const auto N = static_cast<int>(state.range(0));
  using precision = float;
  using OpClass   = cutlass::arch::OpClassSimt;
  using SmArch    = cutlass::arch::Sm80;
  using RingOp = cuasr::or_and<precision>;

  using Srgemm = cuasr::gemm::device::Srgemm<                           //
      RingOp,                                                           //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::ColumnMajor,                   //
      precision, cutlass::layout::RowMajor,
      precision, OpClass, SmArch>;

  // setup bench harness
  cuasr::bench::device::BenchHarness<Srgemm> bench({ N, N, N });

  // benchmark loop
  for (auto _ : state) {
    benchmark::DoNotOptimize(bench.run());
    hipDeviceSynchronize();
  }

  double flops_per_itr = 2.0 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);
}
BENCHMARK(BM_SM80_default_or_and_f32_srgemm_nn_t)
    ->RangeMultiplier(2)->Range(256, 4096);
