#include "hip/hip_runtime.h"
#include "fwgpu/gpu_srgemm.hpp"

#include "cuasr/arch/srmma.h"
#include "cuasr/gemm/device/default_srgemm_configuration.h"
#include "cuasr/gemm/device/srgemm.h"

#include "cutlass/functional.h"

namespace fwgpu {

auto cutlass_srsgemm_nn(
    int M,
    int N,
    int K,
    float const *A,
    int lda,
    float const *B,
    int ldb,
    float *C,
    int ldc,
    float *D,
    bool do_epilogue_min,
    void *stream) -> int {
  hipStream_t stream_ = nullptr;
  if (stream) {
    stream_ = *(static_cast<hipStream_t *>(stream));
  }
  // compile time configuration of this srgemm kernel
  using OperatorClass  = cutlass::arch::OpClassSimt;
  using SmArch         = cutlass::arch::Sm50;
  using TropicalConfig = typename cuasr::gemm::device::DefaultSemiRingConfiguration<
      float, float, float, float,
      cuasr::minimum<float>, cuasr::plus<float>, OperatorClass, SmArch>;

  using AdditionOp       = TropicalConfig::AdditionOp;
  using MultiplicationOp = TropicalConfig::MultiplicationOp;
  using ColumnMajor      = cutlass::layout::ColumnMajor;
  using ThreadblockShape = typename TropicalConfig::ThreadblockShape;
  using WarpShape        = typename TropicalConfig::WarpShape;
  using InstructionShape = typename TropicalConfig::InstructionShape;
  using EpilogueOutputOp = typename TropicalConfig::EpilogueOutputOp;
  using ThreadblockSwizzle =
      typename cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>;
  constexpr int Stages     = TropicalConfig::kStages;
  constexpr int AlignmentA = TropicalConfig::kAlignmentA;
  constexpr int AlignmentB = TropicalConfig::kAlignmentB;

  using cuASR_MinPlus_SGEMM = cuasr::gemm::device::Srgemm<
      AdditionOp,         // Thread level SemiRing operator
      MultiplicationOp,   // Thread level SemiRing operator
      float,              // element type of A
      ColumnMajor,        // layout of A
      float,              // element type of B
      ColumnMajor,        // layout of B
      float,              // element type of C
      ColumnMajor,        // layout of C
      float,              // element type of D
      OperatorClass,      // Logical operator class (SIMT/Tensor)
      SmArch,             // cuda architecture
      ThreadblockShape,   // GEMM shape at CTA level
      WarpShape,          // GEMM shape at Warp level
      InstructionShape,   // GEMM shape at thread level
      EpilogueOutputOp,   // Epilogue operator at thread level
      ThreadblockSwizzle, // GEMM threadblock swizzler
      Stages,             // Pipeline stages for shmem
      AlignmentA,         // Alignment of A elements
      AlignmentB,         // Alignment of B elements
      false               // SplitKSerial
  >;

  float alpha = MultiplicationOp::Identity;
  float beta
      = do_epilogue_min ? MultiplicationOp::Identity : MultiplicationOp::Annihilator;
  // construct kernel arguments struct
  cuASR_MinPlus_SGEMM::Arguments args(
      { M, N, K },    // Problem dimensions
      { A, lda },     // Tensor-ref for source matrix A
      { B, ldb },     // Tensor-ref for source matrix B
      { C, ldc },     // Tensor-ref for source matrix C
      { D, ldc },     // Tensor-ref for destination matrix D
      { alpha, beta } // True if we perform a final min with source matrix C
  );

  // launch SRGEMM kernel
  cuASR_MinPlus_SGEMM minplus_gemm;
  cutlass::Status status = minplus_gemm(args, nullptr, stream_);
  return static_cast<int>(status);
}

auto cutlass_srsgemm_nn(
    int M,
    int N,
    int K,
    float const *A,
    int lda,
    float const *B,
    int ldb,
    float *C,
    int ldc,
    bool do_epilogue_min,
    void *stream) -> int {
  return cutlass_srsgemm_nn(M, N, K, A, lda, B, ldb, C, ldc, C, do_epilogue_min, stream);
}

} // namespace fwgpu
